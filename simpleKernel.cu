#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

// Function to generate the Fibonacci sequence
void generateFibonacci(thrust::host_vector<int> &fibSeq, int size)
{
    fibSeq.resize(size);
    fibSeq[0] = 0;
    fibSeq[1] = 1;

    for (int i = 2; i < size; i++)
    {
        fibSeq[i] = fibSeq[i - 1] + fibSeq[i - 2];
    }
}

// Function to perform manual sorting using bubble sort
void bubbleSort(thrust::host_vector<int> &arr)
{
    int n = arr.size();
    for (int i = 0; i < n - 1; i++)
    {
        for (int j = 0; j < n - i - 1; j++)
        {
            if (arr[j] > arr[j + 1])
            {
                // Swap elements
                int temp = arr[j];
                arr[j] = arr[j + 1];
                arr[j + 1] = temp;
            }
        }
    }
}

// Function to merge two sorted subarrays
void merge(thrust::host_vector<int> &arr, int l, int m, int r)
{
    int n1 = m - l + 1;
    int n2 = r - m;

    // Create temporary vectors for the left and right subarrays
    thrust::host_vector<int> left(n1);
    thrust::host_vector<int> right(n2);

    // Copy data to the temporary vectors
    for (int i = 0; i < n1; i++)
    {
        left[i] = arr[l + i];
    }
    for (int j = 0; j < n2; j++)
    {
        right[j] = arr[m + 1 + j];
    }

    // Merge the temporary arrays back into arr[l..r]
    int i = 0; // Initial index of first subarray
    int j = 0; // Initial index of second subarray
    int k = l; // Initial index of merged subarray

    while (i < n1 && j < n2)
    {
        if (left[i] <= right[j])
        {
            arr[k] = left[i];
            i++;
        }
        else
        {
            arr[k] = right[j];
            j++;
        }
        k++;
    }

    // Copy the remaining elements of left[] if there are any
    while (i < n1)
    {
        arr[k] = left[i];
        i++;
        k++;
    }

    // Copy the remaining elements of right[] if there are any
    while (j < n2)
    {
        arr[k] = right[j];
        j++;
        k++;
    }
}

// Function to perform Merge Sort
void mergeSort(thrust::host_vector<int> &arr, int l, int r)
{
    if (l < r)
    {
        int m = l + (r - l) / 2;

        // Sort the left and right subarrays
        mergeSort(arr, l, m);
        mergeSort(arr, m + 1, r);

        // Merge the sorted subarrays
        merge(arr, l, m, r);
    }
}

// Function to perform manual sorting using QuickSort
void quickSort(thrust::host_vector<int> &arr, int low, int high)
{
    if (low < high)
    {
        // Partition the array
        int pivot = arr[high];
        int i = (low - 1);

        for (int j = low; j <= high - 1; j++)
        {
            if (arr[j] <= pivot)
            {
                i++;
                std::swap(arr[i], arr[j]);
            }
        }
        std::swap(arr[i + 1], arr[high]);

        // Recursively sort the left and right subarrays
        quickSort(arr, low, i);
        quickSort(arr, i + 2, high);
    }
}

// Function to heapify a subtree rooted with node i which is an index in arr[]. n is the size of the heap
void heapify(thrust::host_vector<int> &arr, int n, int i)
{
    int largest = i;   // Initialize largest as root
    int l = 2 * i + 1; // left = 2*i + 1
    int r = 2 * i + 2; // right = 2*i + 2

    // If left child is larger than root
    if (l < n && arr[l] > arr[largest])
        largest = l;

    // If right child is larger than largest so far
    if (r < n && arr[r] > arr[largest])
        largest = r;

    // If largest is not root
    if (largest != i)
    {
        std::swap(arr[i], arr[largest]);

        // Recursively heapify the affected sub-tree
        heapify(arr, n, largest);
    }
}

// Main function to perform heap sort
void heapSort(thrust::host_vector<int> &arr)
{
    int n = arr.size();

    // Build heap (rearrange array)
    for (int i = n / 2 - 1; i >= 0; i--)
        heapify(arr, n, i);

    // One by one extract an element from heap
    for (int i = n - 1; i >= 0; i--)
    {
        // Move current root to end
        std::swap(arr[0], arr[i]);

        // call max heapify on the reduced heap
        heapify(arr, i, 0);
    }
}

// Functor for transforming values for descending sorting using Thrust
struct SortTransform
{
    __host__ __device__ int operator()(int x) const
    {
        return -x; // Negate the values to sort in descending order
    }
};

int main()
{
    int size = 1000;

    // Allocate memory for input and output arrays on the host
    thrust::host_vector<int> h_a(size);

    // Generate the Fibonacci sequence on the host
    generateFibonacci(h_a, size);

    // Print the generated sequence
    printf("Generated Sequence:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a[i]);
    }
    printf("\n\n");

    // Create CUDA events for measuring time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Sort the array manually using bubble sort
    hipEventRecord(start);
    bubbleSort(h_a);
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_bubble = 0;
    hipEventElapsedTime(&milliseconds_bubble, start, stop);

    // Copy the sorted array for later comparison
    thrust::host_vector<int> h_a_bubble = h_a;

    // Sort the array manually using merge sort
    hipEventRecord(start);
    mergeSort(h_a, 0, size - 1);
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_merge = 0;
    hipEventElapsedTime(&milliseconds_merge, start, stop);

    // Copy the sorted array from device to host
    thrust::host_vector<int> h_a_merge = h_a;

    // Sort the array manually using QuickSort
    hipEventRecord(start);
    quickSort(h_a, 0, size - 1);
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_quick = 0;
    hipEventElapsedTime(&milliseconds_quick, start, stop);

    // Sort the array manually using HeapSort
    hipEventRecord(start);
    heapSort(h_a);
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_heap = 0;
    hipEventElapsedTime(&milliseconds_heap, start, stop);

    // Allocate memory for input array on the device
    thrust::device_vector<int> d_a = h_a;

    // Sort the array using Thrust
    hipEventRecord(start);
    thrust::sort(d_a.begin(), d_a.end());
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_thrust = 0;
    hipEventElapsedTime(&milliseconds_thrust, start, stop);

    // Copy the sorted array from device to host
    h_a = d_a;

    // Allocate memory for input array on the device
    thrust::device_vector<int> d_a_transformed = h_a;

    // Transform the array using Thrust
    thrust::transform(d_a_transformed.begin(), d_a_transformed.end(), d_a_transformed.begin(), SortTransform());

    // Sort the transformed array using Thrust
    hipEventRecord(start);
    thrust::sort(d_a_transformed.begin(), d_a_transformed.end());
    hipEventRecord(stop);

    // Synchronize events and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds_thrust_transform = 0;
    hipEventElapsedTime(&milliseconds_thrust_transform, start, stop);

    // Copy the sorted array from device to host
    h_a = d_a_transformed;

    // Print the sorted arrays
    printf("Bubble Sort:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a_bubble[i]);
    }
    printf("\n");

    printf("Merge Sort:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a_merge[i]);
    }
    printf("\n");

    printf("Quick Sort:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    printf("Heap Sort:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    printf("Thrust Sort:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", d_a[i]);
    }
    printf("\n");

    printf("Thrust Sort + Transformation:\n");
    for (int i = 0; i < size; i++)
    {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    // Print the elapsed times
    printf("\n");
    printf("\n");
    printf("\n");

    printf("~ Manual Sorting Algorithms :");
    printf("\n");
    printf("Bubble Sort Time: %.3f ms\n", milliseconds_bubble);
    printf("Quick Sort Time: %.3f ms\n", milliseconds_quick);
    printf("Merge Sort Time: %.3f ms\n", milliseconds_merge);
    printf("Heap Sort Time: %.3f ms\n", milliseconds_heap);
    printf("\n");
    printf("\n");
    printf("~ Sorting by Thrust library : ");
    printf("\n");
    printf("Thrust Sort Time: %.3f ms\n", milliseconds_thrust);
    printf("\n");
    printf("~Thrust Sorting Algorithm : ");
    printf("\n");
    printf("Thrust Sort + Transformation Time: %.3f ms\n", milliseconds_thrust_transform);
    printf("\n");
    printf("\n");

    return 0;
}
